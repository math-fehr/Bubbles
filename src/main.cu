#include "hip/hip_runtime.h"
#include "assert_cuda.hpp"
#include "geom.h"
#include "interop_window.hpp"
#include "kernel.cuh"
#include "light.h"
#include "object.h"
#include <cuda_gl_interop.h>
#include <random>

using namespace std;

static void show_fps_and_window_size(GLFWwindow *window) {
  // fps counter in static variables
  static double previous_time = 0.0;
  static int frame_count = 0;

  const double current_time = glfwGetTime();
  const double elapsed = current_time - previous_time;

  if (elapsed > 0.5) {
    previous_time = current_time;

    const double fps = (double)frame_count / elapsed;

    int width, height;
    glfwGetFramebufferSize(window, &width, &height);

    char tmp[64];
    sprintf(tmp, "(%u x %u) - FPS: %.2f", width, height, fps);

    glfwSetWindowTitle(window, tmp);

    frame_count = 0;
  }

  frame_count++;
}

void add_scene_box(std::vector<Object> &objects) {
  Object object;
  Texture texture;
  texture.type = TextureType::checkboard;
  texture.checkboard.color1 = white;
  texture.checkboard.color2 = white * 0.5;
  texture.checkboard.n_subdivision = 10.0f;
  texture.factors = Factors::opaque(0.7f);
  Vec3f min_pos = Vec3f{-31.0f, -31.0f, -31.0f};
  Vec3f max_pos = Vec3f{31.0f, 31.0f, 31.0f};
  object.texture = texture;
  object.type = ObjectType::box;
  object.box = Box(min_pos, max_pos);
  objects.push_back(object);
}

void add_mega_scene_box(std::vector<Object> &objects) {
  Object object;
  Texture texture;
  texture.type = TextureType::uniform_color;
  texture.uniform_color.color = black;
  texture.factors = Factors::opaque(0.0f);
  Vec3f min_pos = Vec3f{-3100000.0f, -3100000.0f, -3100000.0f};
  Vec3f max_pos = Vec3f{3100000.0f, 3100000.0f, 3100000.0f};
  object.texture = texture;
  object.type = ObjectType::box;
  object.box = Box(min_pos, max_pos);
  objects.push_back(object);
}

void update_camera(Camera &camera, const InteropWindow &win, real time) {
  real speed = 5.0; // in unit per second
  glfwPollEvents();
  if (glfwGetKey(win.window.get(), GLFW_KEY_LEFT_SHIFT) == GLFW_PRESS)
    speed = 10.0;
  real d = speed * time;
  if (glfwGetKey(win.window.get(), GLFW_KEY_W) == GLFW_PRESS)
    camera.move_front(d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_S) == GLFW_PRESS)
    camera.move_front(-d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_A) == GLFW_PRESS)
    camera.move_lat(-d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_D) == GLFW_PRESS)
    camera.move_lat(d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_SPACE) == GLFW_PRESS)
    camera.move_up(d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_LEFT_CONTROL) == GLFW_PRESS)
    camera.move_up(-d);
}

Vec3f gradient(const function<real(Vec3f)> &f, Vec3f pos) {
  real eps = 1e-4;
  return Vec3f{(f(pos + eps * X) - f(pos - eps * X)) / (2 * eps),
               (f(pos + eps * Y) - f(pos - eps * Y)) / (2 * eps),
               (f(pos + eps * Z) - f(pos - eps * Z)) / (2 * eps)};
}

int main(int argc, char *argv[]) {
  std::vector<Object> objects;
  objects.reserve(100);

  add_mega_scene_box(objects);
  add_scene_box(objects);

  objects.push_back(
      Object(Box{Vec3f{-10, -10, -10}, Vec3f{-5, -5, -5}})
          .set(Texture(UniformColor{red}).set(Factors::opaque(0.7f, 0.5, 5))));

  // metal shininess 1000, specular 0.5

  objects.push_back(Object(Pipe{Vec3f{0.f, -10.f, 0.f}})
                        .set(Texture(WoodTexture{}).set(Factors::opaque(0.8))));

  objects.push_back(
      Object(FutureBubble(objects.back().pipe))
      .set(Texture(BubbleTexture{5.0}).set(Factors::full(0.6, 20, 500, 0.1, 0.8, 1.005))));

  int bubbles_start = objects.size();

  for (int i = 0; i < MAX_NUM_BUBBLES; ++i) {
    objects.push_back(
        Object(Bubble{Vec3f{2 * (i % 20) - 20.f, 2.f * (i / 20), 10}, 0.2, 0.1})
            .set(Texture(BubbleTexture{5.0})
                     .set(Factors::full(0.6, 20, 500, 0.1, 0.8, 1.005))));

    objects.back().speed = Vec3f{0, 0, -1};
  }

  Object *d_objects = nullptr;
  cuda(Malloc(&d_objects, sizeof(Object) * MAX_OBJECTS));
  cuda(Memcpy(d_objects, objects.data(), sizeof(Object) * objects.size(),
              hipMemcpyHostToDevice));

  PointLight light{Vec3f{-30.0f, 0.0f, 0.0f}, Color{1.0f, 1.0f, 1.0f}};
  AmbiantLight ambiant_light{1.0f, 1.0f, 1.0f};

  Scene scene{d_objects, (unsigned)objects.size(), light, ambiant_light};

  unsigned init_width = X_BASE_SIZE;
  unsigned init_height = Y_BASE_SIZE;

  Vec3f camera_pos{3.0f, -10.0f, 0.0f};
  Vec3f camera_dir{-1.f, 0, 0};
  Vec3f camera_up{0, 1, 0};

  Camera camera(camera_pos, camera_dir, camera_up, 51.52f * M_PI / 180.0f,
                init_width, init_height);

  camera.gamma = 1.5;

  InteropWindow interop_window(init_width, init_height);

  interop_window.cursor_callback = [&camera](GLFWwindow *, double xupd,
                                             double yupd) {
    camera.rotate_lat(xupd * 0.0005);
    camera.rotate_up(-yupd * 0.0005);
  };

  double time = glfwGetTime();
  double lasttime = glfwGetTime();

  // Main loop
  while (!glfwWindowShouldClose(interop_window.window.get())) {
    show_fps_and_window_size(interop_window.window.get());

    // Execute the CUDA code
    std::tie(camera.screen_width, camera.screen_height) =
        interop_window.interop_data.get_size();

    kernel_launcher(interop_window.interop_data.get_current_cuda_array(), scene,
                    camera);

    // Event management
    time = glfwGetTime();
    update_camera(camera, interop_window, time - lasttime);
    real delta_time = time - lasttime;

    // update physics, simulation, ...

    Pipe pipe;
    for (int j = 0; j < objects.size(); ++j) {
      if (objects[j].type == ObjectType::pipe) {
        pipe = objects[j].pipe;
        break;
      }
    }

    // Update growing bubble
    for (int i = 0; i < objects.size(); ++i) {
      if (objects[i].type != ObjectType::future_bubble) {
        continue;
      }

      FutureBubble &bubble = objects[i].future_bubble;
      real radius = bubble.compute_radius(pipe);
      if (bubble.touch_hole) {
        // If we finish growing
        if (radius > bubble.stop_radius && radius > bubble.radius) {
          Bubble new_bubble = bubble.transform();
          objects[i].type = ObjectType::bubble;
          objects[i].bubble = new_bubble;
          objects[i].speed = Vec3f{0,1,0};
        } else {
          bubble.center.y += delta_time;
          bubble.set_radius(radius);
        }
      } else {
        bubble.center.y += delta_time * 0.1;
        bubble.limit_plane += delta_time * 0.1;
        if (radius < bubble.radius) {
          bubble.touch_hole = true;
          bubble.set_radius(radius);
          bubble.set_limit_plane(pipe);
        }
      }
    }

    // Maybe add a new bubble
    bool has_future_bubble = false;
    for(int i = 0; i < objects.size(); ++i) {
      if(objects[i].type == ObjectType::future_bubble) {
        has_future_bubble = true;
        break;
      }
    }
    if(objects.size() < (MAX_OBJECTS - 1) && has_future_bubble == false) {
      objects.push_back(Object(FutureBubble(pipe))
                        .set(Texture(BubbleTexture{5.0}).set(Factors::full(0.6, 20, 500, 0.1, 0.8, 1.005))));
      scene.n_objects++;
    }


    // Update bubbles
    random_device dev;
    normal_distribution<real> dist(0, 0.5);
    real k = 0.1;
    real bubble_mass = 1;

    for (int i = 0; i < objects.size(); ++i) {
      if(objects[i].type != ObjectType::bubble) {
        continue;
      }

      Vec3f grad{0, 0, 0};
      grad += -gradient(
          [&](Vec3f p) {
            return exp(sqrtf(objects[i].sphere.radius2) + objects[1].sdf(p));
          },
          objects[i].pos());
      for (int j = 2; j < objects.size(); ++j) {
        if (j == i) continue;
        grad += -gradient(
            [&](Vec3f p) {
              return exp(sqrtf(objects[i].sphere.radius2) - objects[j].sdf(p));
            },
            objects[i].pos());
      }
      grad += -gradient(
          [&](Vec3f p) { return exp(2 - (p - camera.get_pos()).norm()); },
          objects[i].pos());

      Vec3f force =
          -k * objects[i].speed + Vec3f{dist(dev), dist(dev), dist(dev)} + grad;

      Vec3f accel = force / bubble_mass;

      objects[i].speed += accel * (time - lasttime);
    }

    for (int i = 0; i < objects.size(); ++i) {
      if(objects[i].type == ObjectType::bubble) {
        objects[i].move(sqrtf(objects[i].sphere.radius2) * objects[i].speed *
                        (time - lasttime));
      }
    }

    lasttime = time;

    // Wait for GPU to finish rendering
    hipDeviceSynchronize();
    cuda(Memcpy(d_objects, objects.data(), sizeof(Object) * objects.size(),
                hipMemcpyHostToDevice));

    // Switch buffers
    interop_window.interop_data.blit_buffer();
    interop_window.interop_data.change_buffer();
    glfwSwapBuffers(interop_window.window.get());
  }

  return EXIT_SUCCESS;
}
