#include "assert_cuda.hpp"
#include "geom.h"
#include "interop_window.hpp"
#include "kernel.cuh"
#include "object.h"
#include <cuda_gl_interop.h>

static void show_fps_and_window_size(GLFWwindow *window) {
  // fps counter in static variables
  static double previous_time = 0.0;
  static int frame_count = 0;

  const double current_time = glfwGetTime();
  const double elapsed = current_time - previous_time;

  if (elapsed > 0.5) {
    previous_time = current_time;

    const double fps = (double)frame_count / elapsed;

    int width, height;
    glfwGetFramebufferSize(window, &width, &height);

    char tmp[64];
    sprintf(tmp, "(%u x %u) - FPS: %.2f", width, height, fps);

    glfwSetWindowTitle(window, tmp);

    frame_count = 0;
  }

  frame_count++;
}

int main(int argc, char *argv[]) {

  std::vector<Object> objects;
  for (float i = 0.f; i < 19.99f; i += 2.0f) {
    Vec3f pos{i - 10.0f, i - 10.0f, -10.0f};
    float radius = 1.f;
    Color color{1.0f, 1.0f, 1.0f};
    Object object;
    object.color = color;
    object.type = ObjectType::sphere;
    object.sphere = Sphere{pos, radius};
    objects.push_back(object);
  }

  Object *d_objects = nullptr;
  cuda(Malloc(&d_objects, sizeof(Object) * objects.size()));
  cuda(Memcpy(d_objects, objects.data(), sizeof(Object) * objects.size(),
              hipMemcpyHostToDevice));

  unsigned init_width = 1024;
  unsigned init_height = 720;

  Vec3f camera_pos{0.0f, 0.0f, 20.0f};
  Vec3f camera_dir{0, 0, -1};
  Vec3f camera_up{0, 1, 0};
  // Vec3f camera_to_world_x{1.0f, 0.0f, 0.0f};
  // Vec3f camera_to_world_y{0.0f, 1.0f, 0.0f};
  // Vec3f camera_to_world_z{0.0f, 0.0f, 1.0f};
  // Mat3f
  // camera_to_world{camera_to_world_x,camera_to_world_y,camera_to_world_z};

  Camera camera(camera_pos, camera_dir, camera_up, 51.52f * M_PI / 180.0f,
                init_width, init_height);

  InteropWindow interop_window(init_width, init_height);

  interop_window.key_callbacks.insert(
      {GLFW_KEY_W, [&camera](GLFWwindow *, int action, int mods) {
         camera.move_front(0.1);
       }});
  interop_window.key_callbacks.insert(
      {GLFW_KEY_S, [&camera](GLFWwindow *, int action, int mods) {
         camera.move_front(-0.1);
       }});
  interop_window.key_callbacks.insert(
      {GLFW_KEY_A, [&camera](GLFWwindow *, int action, int mods) {
         camera.move_lat(-0.1);
       }});
  interop_window.key_callbacks.insert(
      {GLFW_KEY_D, [&camera](GLFWwindow *, int action, int mods) {
         camera.move_lat(0.1);
       }});
  interop_window.key_callbacks.insert(
      {GLFW_KEY_SPACE,
       [&camera](GLFWwindow *, int action, int mods) { camera.move_up(0.1); }});
  interop_window.key_callbacks.insert(
      {GLFW_KEY_LEFT_CONTROL, [&camera](GLFWwindow *, int action, int mods) {
         camera.move_up(-0.1);
       }});
  interop_window.cursor_callback = [&camera](GLFWwindow *, double xupd,
                                             double yupd) {
    camera.rotate_lat(xupd * 0.005);
    camera.rotate_up(-yupd * 0.005);
  };

  // Main loop
  while (!glfwWindowShouldClose(interop_window.window.get())) {
    show_fps_and_window_size(interop_window.window.get());

    // Execute the CUDA code
    std::tie(camera.screen_width, camera.screen_height) =
        interop_window.interop_data.get_size();
    kernel_launcher(interop_window.interop_data.get_current_cuda_array(),
                    d_objects, objects.size(), camera);

    // Switch buffers
    interop_window.interop_data.blit_buffer();
    interop_window.interop_data.change_buffer();
    glfwSwapBuffers(interop_window.window.get());

    // Get events
    glfwPollEvents();
  }

  exit(EXIT_SUCCESS);
}
