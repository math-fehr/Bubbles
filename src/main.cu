#include "assert_cuda.hpp"
#include "geom.h"
#include "interop_window.hpp"
#include "kernel.cuh"
#include "light.h"
#include "object.h"
#include <cuda_gl_interop.h>

using namespace std;

static void show_fps_and_window_size(GLFWwindow *window) {
  // fps counter in static variables
  static double previous_time = 0.0;
  static int frame_count = 0;

  const double current_time = glfwGetTime();
  const double elapsed = current_time - previous_time;

  if (elapsed > 0.5) {
    previous_time = current_time;

    const double fps = (double)frame_count / elapsed;

    int width, height;
    glfwGetFramebufferSize(window, &width, &height);

    char tmp[64];
    sprintf(tmp, "(%u x %u) - FPS: %.2f", width, height, fps);

    glfwSetWindowTitle(window, tmp);

    frame_count = 0;
  }

  frame_count++;
}

void add_scene_box(std::vector<Object> &objects) {
  Object object;
  Texture texture;
  texture.type = TextureType::checkboard;
  texture.checkboard.color1 = white;
  texture.checkboard.color2 = white*0.5;
  texture.checkboard.n_subdivision = 10.0f;
  texture.factors = Factors::opaque(0.7f);
  Vec3f min_pos = Vec3f{-31.0f, -31.0f, -31.0f};
  Vec3f max_pos = Vec3f{31.0f, 31.0f, 31.0f};
  object.texture = texture;
  object.type = ObjectType::box;
  object.box = Box(min_pos, max_pos);
  objects.push_back(object);
}

void add_mega_scene_box(std::vector<Object> &objects) {
  Object object;
  Texture texture;
  texture.type = TextureType::uniform_color;
  texture.uniform_color.color = black;
  texture.factors = Factors::opaque(0.0f);
  Vec3f min_pos = Vec3f{-3100000.0f, -3100000.0f, -3100000.0f};
  Vec3f max_pos = Vec3f{3100000.0f, 3100000.0f, 3100000.0f};
  object.texture = texture;
  object.type = ObjectType::box;
  object.box = Box(min_pos, max_pos);
  objects.push_back(object);
}


void update_camera(Camera &camera, const InteropWindow &win, real time) {
  real speed = 5.0; // in unit per second
  glfwPollEvents();
  if (glfwGetKey(win.window.get(), GLFW_KEY_LEFT_SHIFT) == GLFW_PRESS)
    speed = 50.0;
  real d = speed * time;
  if (glfwGetKey(win.window.get(), GLFW_KEY_W) == GLFW_PRESS)
    camera.move_front(d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_S) == GLFW_PRESS)
    camera.move_front(-d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_A) == GLFW_PRESS)
    camera.move_lat(-d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_D) == GLFW_PRESS)
    camera.move_lat(d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_SPACE) == GLFW_PRESS)
    camera.move_up(d);
  if (glfwGetKey(win.window.get(), GLFW_KEY_LEFT_CONTROL) == GLFW_PRESS)
    camera.move_up(-d);
}

int main(int argc, char *argv[]) {
  std::vector<Object> objects;
  add_scene_box(objects);
  add_mega_scene_box(objects);

  // The bubbly bubble
  Object object;
  Vec3f pos{10.0f, 10.0f, 10.0f};
  object.texture.type = TextureType::bubble;
  object.texture.factors = Factors::full(0.0, 0.0, 0.7, 1.01);
  object.type = ObjectType::sphere;
  object.sphere = Sphere{pos, 1.0f};
  objects.push_back(object);

  pos = Vec3f{-30,1,0};
  object.texture.type = TextureType::checkboard;
  object.texture.factors = Factors::opaque(0.6f);
  object.texture.checkboard.color1 = Color{1.0f, 1.0f, 1.0f};
  object.texture.checkboard.color2 = Color{0.0f, 0.0f, 0.0f};
  object.texture.checkboard.n_subdivision = 5.0f;
  object.type = ObjectType::sphere;
  object.sphere = Sphere{pos, 0.1f};
  objects.push_back(object);

  pos = Vec3f{-30,-1,0};
  object.texture.type = TextureType::checkboard;
  object.texture.factors = Factors::opaque(0.6f);
  object.texture.checkboard.color1 = Color{1.0f, 1.0f, 1.0f};
  object.texture.checkboard.color2 = Color{0.0f, 0.0f, 0.0f};
  object.texture.checkboard.n_subdivision = 5.0f;
  object.type = ObjectType::sphere;
  object.sphere = Sphere{pos, 0.1};
  objects.push_back(object);


  Texture texture;
  texture.type = TextureType::uniform_color;
  texture.uniform_color.color = red;
  texture.factors = Factors::full(0.7, 0.1,0.8,1.5);
  object.texture = texture;
  // object.type = ObjectType::box2;
  // object.box2 = Boxv2(Vec3f{-10,-10,-10},5*X,5*Y,5*Z);
  object.type = ObjectType::box;
  object.box = Box{Vec3f{-10,-10,-10},Vec3f{-5,-5,-5}};

  objects.push_back(object);

  Object *d_objects = nullptr;
  cuda(Malloc(&d_objects, sizeof(Object) * objects.size()));
  cuda(Memcpy(d_objects, objects.data(), sizeof(Object) * objects.size(),
              hipMemcpyHostToDevice));

  PointLight light{Vec3f{-30.0f, 0.0f, 0.0f}, Color{1.0f, 1.0f, 1.0f}};
  AmbiantLight ambiant_light{1.0f, 1.0f, 1.0f};

  Scene scene{d_objects, (unsigned)objects.size(), light, ambiant_light};

  unsigned init_width = X_BASE_SIZE;
  unsigned init_height = Y_BASE_SIZE;

  Vec3f camera_pos{10.0f, 10.0f, 10.0f};
  Vec3f camera_dir{1, 1, 1};
  Vec3f camera_up{0, 1, 0};

  Camera camera(camera_pos, camera_dir, camera_up, 51.52f * M_PI / 180.0f,
                init_width, init_height);

  camera.gamma = 1.5;

  InteropWindow interop_window(init_width, init_height);

  interop_window.cursor_callback = [&camera](GLFWwindow *, double xupd,
                                             double yupd) {
    camera.rotate_lat(xupd * 0.0005);
    camera.rotate_up(-yupd * 0.0005);
  };

  double time = glfwGetTime();
  double lasttime = glfwGetTime();

  // Main loop
  while (!glfwWindowShouldClose(interop_window.window.get())) {
    show_fps_and_window_size(interop_window.window.get());

    // Execute the CUDA code
    std::tie(camera.screen_width, camera.screen_height) =
        interop_window.interop_data.get_size();

    kernel_launcher(interop_window.interop_data.get_current_cuda_array(), scene,
                    camera);

    // Event management
    time = glfwGetTime();
    update_camera(camera, interop_window, time - lasttime);
    lasttime = time;

    // update physics, simulation, ...

    // Wait for GPU to finish rendering
    hipDeviceSynchronize();

    // Switch buffers
    interop_window.interop_data.blit_buffer();
    interop_window.interop_data.change_buffer();
    glfwSwapBuffers(interop_window.window.get());
  }

  exit(EXIT_SUCCESS);
}
