#include "assert_cuda.hpp"
#include "interop_window.hpp"
#include "kernel.cuh"
#include <cuda_gl_interop.h>
#include "geom.h"
#include "object.h"

static void show_fps_and_window_size(GLFWwindow *window) {
  // fps counter in static variables
  static double previous_time = 0.0;
  static int frame_count = 0;

  const double current_time = glfwGetTime();
  const double elapsed = current_time - previous_time;

  if (elapsed > 0.5) {
    previous_time = current_time;

    const double fps = (double)frame_count / elapsed;

    int width, height;
    glfwGetFramebufferSize(window, &width, &height);

    char tmp[64];
    sprintf(tmp, "(%u x %u) - FPS: %.2f", width, height, fps);

    glfwSetWindowTitle(window, tmp);

    frame_count = 0;
  }

  frame_count++;
}

int main(int argc, char *argv[]) {

  std::vector<Object> objects;
  for(float i = 0.f; i < 9.99f; i+=0.1f) {
    Vec3f pos{i - 0.5f, i - 0.5f, i - 0.5f};
    float radius = i*0.1f;
    Color color{i/10, i/10, i/10};
    Object object;
    object.color = color;
    object.type = ObjectType::sphere;
    object.sphere = Sphere{pos,radius};
    objects.push_back(object);
  }
  Object object;
  object.color = Color{1.0f,0.0f,0.0f};
  object.type = ObjectType::plane;
  object.plane = Plane(Vec3f{1.0f,0.0f,0.0f}, 100.0f);
  objects.push_back(object);

  Object* d_objects = nullptr;
  cuda(Malloc(&d_objects, sizeof(Object) * objects.size()));
  cuda(Memcpy(d_objects, objects.data(), sizeof(Object) * objects.size(), hipMemcpyHostToDevice));

  InteropWindow interop_window(640, 480);

  // Main loop
  while (!glfwWindowShouldClose(interop_window.window.get())) {
    show_fps_and_window_size(interop_window.window.get());

    // Execute the CUDA code
    unsigned width, height;
    std::tie(width, height) = interop_window.interop_data.get_size();
    kernel_launcher(interop_window.interop_data.get_current_cuda_array(), width,
                    height, d_objects, objects.size());

    // Switch buffers
    interop_window.interop_data.blit_buffer();
    interop_window.interop_data.change_buffer();
    glfwSwapBuffers(interop_window.window.get());

    // Get events
    glfwPollEvents();
  }

  exit(EXIT_SUCCESS);
}
