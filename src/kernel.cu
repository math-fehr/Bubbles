#include "hip/hip_runtime.h"
#include "assert_cuda.hpp"
#include "kernel.cuh"

#include "camera.h"
#include "geom.h"
#include "light.h"
#include "object.h"

#define THREADS_PER_BLOCK 256

// The surface where CUDA will write
surface<void, cudaSurfaceType2D> surf;

// The value of a pixel
struct RGBA {
  unsigned r : 8;
  unsigned g : 8;
  unsigned b : 8;
  unsigned a : 8;
};

__device__ real intersect(Object object, Rayf ray) {
  if (object.type == ObjectType::sphere) {
    return object.sphere.inter(ray);
  } else if (object.type == ObjectType::plane) {
    return object.plane.inter(ray);
  } else if (object.type == ObjectType::box) {
    return object.box.inter(ray);
  } else {
    return -1.f;
  }
}

__device__ Vec3f normal(Object object, Rayf ray, real intersection_distance) {
  if (object.type == ObjectType::sphere) {
    return object.sphere.normal(ray(intersection_distance));
  } else if (object.type == ObjectType::plane) {
    return object.plane.normal(ray);
  } else if (object.type == ObjectType::box) {
    return object.box.normal(ray, ray(intersection_distance));
  } else {
    return Vec3f{};
  }
}

struct Intersection {
  int object_id;
  Object object;
  real distance;
  Vec3f point;
  Vec3f normal;
};

__device__ Intersection intersect_all(Object *objects, unsigned n_objects,
                                      Rayf ray) {
  int front_object = -1;
  real intersection_point = 1.f / 0.f;

  for (int i = 0; i < n_objects; ++i) {
    float intersection_i = intersect(objects[i], ray);
    if (intersection_i > 0.f && intersection_i < intersection_point) {
      intersection_point = intersection_i;
      front_object = i;
    }
  }

  return Intersection{front_object, objects[front_object], intersection_point,
                      ray(intersection_point),
                      normal(objects[front_object], ray, intersection_point)};
}

__device__ Color compute_phong_color(Object *objects, unsigned n_objects,
                                     PointLight light,
                                     AmbiantLight ambiant_light,
                                     Intersection intersection) {

  Color ambiant_color = intersection.object.texture.phong.color *
                        ambiant_light.color *
                        intersection.object.texture.phong.ambiant_factor;

  Rayf light_ray = light.ray_to_point(intersection.point);
  Intersection light_intersection =
      intersect_all(objects, n_objects, light_ray);
  bool light_touch = intersection.object_id == light_intersection.object_id;
  light_touch &= (intersection.point - light_intersection.point).norm() < 1e-3;

  if (!light_touch) {
    return ambiant_color;
  }
  real diffusion_factor = intersection.normal | light_ray.dir;
  diffusion_factor = max(0.0f, min(1.0f, diffusion_factor));
  diffusion_factor *= intersection.object.texture.phong.diffusion_factor;
  Color diffuse_color =
      intersection.object.texture.phong.color * diffusion_factor * light.color;
  return diffuse_color + ambiant_color;
}

__device__ Color compute_texture(Object *objects, unsigned n_objects,
                                 PointLight light, AmbiantLight ambiant_light,
                                 Intersection intersection) {
  switch (intersection.object.texture.type) {
  case TextureType::phong:
    return compute_phong_color(objects, n_objects, light, ambiant_light,
                               intersection);
  }
  return Color{1, 1, 1};
}

/**
 * Entry CUDA kernel. This is the code for one pixel
 */
__global__ void kernel(int counter, Object *objects, unsigned n_objects,
                       Camera camera) {
  PointLight light{Vec3f{-30.0f, 0.0f, 0.0f}, Color{1, 1, 1}};
  AmbiantLight ambiant_light{Color{1.0f, 1.0f, 1.0f}};

  // pixel coordinates
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
  int x_pixel = idx % camera.screen_width;
  int y_pixel = idx / camera.screen_width;

  Rayf ray = camera.get_ray(x_pixel, y_pixel);

  RGBA rgbx;
  rgbx.r = 0, rgbx.g = 0, rgbx.b = 0;

  Intersection intersection = intersect_all(objects, n_objects, ray);
  Rayf light_ray = light.ray_to_point(intersection.point);

  Vec3f normal_vec = normal(intersection.object, ray, intersection.distance);

  Color color = compute_phong_color(objects, n_objects, light, ambiant_light,
                                    intersection);
  rgbx.r = color.r * 255;
  rgbx.g = color.g * 255;
  rgbx.b = color.b * 255;

  if (idx < camera.screen_height * camera.screen_width) {
    surf2Dwrite(rgbx, surf, x_pixel * sizeof(rgbx), y_pixel,
                hipBoundaryModeZero);
  }
}

void kernel_launcher(hipArray_const_t array, Object *objects,
                     unsigned n_objects, Camera camera) {
  // Count the number of frames displayed
  static unsigned counter = 0;
  counter += 1;

  cuda(BindSurfaceToArray(surf, array));

  const int blocks =
      (camera.screen_width * camera.screen_height + THREADS_PER_BLOCK - 1) /
      THREADS_PER_BLOCK;

  if (blocks > 0) {
    kernel<<<blocks, THREADS_PER_BLOCK>>>(counter, objects, n_objects, camera);
  }
}
